
#include <hip/hip_runtime.h>
#ifdef PRECICE_WITH_CUDA

#include <cublas_v2.h>
#include <cuda.h>
#include <cuda_runtime.h>
#include <cuda_runtime_api.h>
#include <cusolverDn.h>
#include <ginkgo/ginkgo.hpp>
#include "device_launch_parameters.h"
#include "mapping/device/CudaQRSolver.cuh"
#include "utils/Event.hpp"
#include "utils/EventUtils.hpp"


void computeQRDecompositionCuda(const int deviceId, const std::shared_ptr<gko::Executor> &exec, gko::matrix::Dense<> *A_Q, gko::matrix::Dense<> *R)
{
  int backupDeviceId{};
  cudaGetDevice(&backupDeviceId);
  cudaSetDevice(deviceId);

  void *dWork{};
  int  *devInfo{};

  // Allocating important CUDA variables
  cudaMalloc((void **) &dWork, sizeof(double));
  cudaMalloc((void **) &devInfo, sizeof(int));

  cusolverDnHandle_t solverHandle;
  cusolverDnCreate(&solverHandle);
  // NOTE: It's important to transpose since cuSolver assumes column-major memory layout
  // Making a copy since every value will be overridden
  // auto A_T = gko::share(gko::matrix::Dense<>::create(exec, gko::dim<2>(A_Q->get_size()[1], A_Q->get_size()[0])));
  // A_Q->transpose(gko::lend(A_T));

  // Setting dimensions for solver
  const unsigned int M = A_Q->get_size()[1];
  const unsigned int N = A_Q->get_size()[0];

  const int lda = max(1, M);
  const int k   = min(M, N);

  size_t dLwork = 0;
  // size_t hLwork = 0;

  // double *dTau{};
  // cudaMalloc((void **) &dTau, sizeof(double) * M);

  // PRECICE_ASSERTs collide with cuda for some (non-extensively investigated) reason

  cublasFillMode_t uplo           = CUBLAS_FILL_MODE_UPPER;
  cusolverStatus_t cusolverStatus = cusolverDnDpotrf_bufferSize(solverHandle, uplo, M, A_Q->get_values(), lda, (int *) &dLwork);
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);

  cudaError_t cudaErrorCode = cudaMalloc((void **) &dWork, sizeof(double) * dLwork);
  assert(cudaSuccess == cudaErrorCode);

  // void *hWork{};

  cusolverStatus = cusolverDnDpotrf(solverHandle, uplo, M, A_Q->get_values(), lda, (double *) dWork, (int) dLwork, devInfo);
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);

  cudaErrorCode = cudaDeviceSynchronize();
  assert(cudaSuccess == cudaErrorCode);

  // Copy A_T to R s.t. the upper triangle corresponds to R
  A_Q->transpose(gko::lend(R));

  // Compute Q
  // cusolverStatus = cusolverDnDorgqr(solverHandle, M, N, k, A_T->get_values(), lda, dTau, (double *) dWork, dLwork, devInfo);
  // assert(cudaSuccess == cudaErrorCode);

  // R->transpose(gko::lend(A_Q));

  cudaDeviceSynchronize();

  // Free the utilizted memory
  // cudaFree(dTau);
  cudaFree(dWork);
  cudaFree(devInfo);
  cusolverDnDestroy(solverHandle);

  // ...and switch back to the GPU used for all coupled solvers
  cudaSetDevice(backupDeviceId);
}
#endif
