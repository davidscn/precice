#include "hip/hip_runtime.h"
#ifndef PRECICE_NO_GINKGO

#include <cmath>
#include <stdio.h>

#include "mapping/impl/DeviceBasisFunctions.cuh"

#define SHARED_HOST_DEVICE_FUNCTION __host__ __device__
#define NUMERICAL_ZERO_DIFFERENCE 1.0e-14

namespace precice {
namespace mapping {

SHARED_HOST_DEVICE_FUNCTION double ThinPlateSplinesFunctor::operator()(const double radius, const std::array<double, 3> params) const
{
  // We don't need to read any values from params since there is no need here
  return std::log(std::max(radius, NUMERICAL_ZERO_DIFFERENCE)) * std::pow(radius, 2);
}

SHARED_HOST_DEVICE_FUNCTION double MultiQuadraticsFunctor::operator()(const double radius, const std::array<double, 3> params) const
{
  double cPow2 = params.at(0);
  return std::sqrt(cPow2 + std::pow(radius, 2));
}

SHARED_HOST_DEVICE_FUNCTION double InverseMultiquadricsFunctor::operator()(const double radius, const std::array<double, 3> params) const
{
  double cPow2 = params.at(0);
  return 1.0 / std::sqrt(cPow2 + std::pow(radius, 2));
}

SHARED_HOST_DEVICE_FUNCTION double VolumeSplinesFunctor::operator()(const double radius, const std::array<double, 3> params) const
{
  return std::abs(radius);
}

SHARED_HOST_DEVICE_FUNCTION double GaussianFunctor::operator()(const double radius, std::array<double, 3> params) const
{
  double shape         = params.at(0);
  double supportRadius = params.at(1);
  double deltaY        = params.at(2);

  if (radius > supportRadius) {
    return 0.0;
  } else {
    return std::exp(-std::pow(shape * radius, 2)) - deltaY;
  }
}

SHARED_HOST_DEVICE_FUNCTION double CompactThinPlateSplinesC2Functor::operator()(const double radius, const std::array<double, 3> params) const
{
  double       r_inv = params.at(0);
  double const p     = radius * r_inv;
  if (p >= 1) {
    return 0.0;
  } else {
    return 1.0 - 30.0 * std::pow(p, 2) - 10.0 * std::pow(p, 3) + 45.0 * std::pow(p, 4) - 6.0 * std::pow(p, 5) - std::pow(p, 3) * 60.0 * std::log(std::max(p, NUMERICAL_ZERO_DIFFERENCE));
  }
}

SHARED_HOST_DEVICE_FUNCTION double CompactPolynomialC0Functor::operator()(const double radius, const std::array<double, 3> params) const
{
  double       r_inv = params.at(0);
  double const p     = radius * r_inv;
  if (p >= 1) {
    return 0.0;
  } else {
    return std::pow(1.0 - p, 2);
  }
}

SHARED_HOST_DEVICE_FUNCTION double CompactPolynomialC2Functor::operator()(const double radius, const std::array<double, 3> params) const
{
  double       r_inv = params.at(0);
  double const p     = radius * r_inv;
  if (p >= 1) {
    return 0.0;
  } else {
    return std::pow(1.0 - p, 4) * (4 * p + 1);
  }
}

SHARED_HOST_DEVICE_FUNCTION double CompactPolynomialC4Functor::operator()(const double radius, const std::array<double, 3> params) const
{
  double       r_inv = params.at(0);
  double const p     = radius * r_inv;
  if (p >= 1) {
    return 0.0;
  } else {
    return std::pow(1.0 - p, 6) * (35 * std::pow(p, 2) + 18 * p + 3, 2);
  }
}

SHARED_HOST_DEVICE_FUNCTION double CompactPolynomialC6Functor::operator()(const double radius, const std::array<double, 3> params) const
{
  double       r_inv = params.at(0);
  double const p     = radius * r_inv;
  if (p >= 1) {
    return 0.0;
  } else {
    return std::pow(1.0 - p, 8) * (32.0 * std::pow(p, 3) + 25.0 * std::pow(p, 2) + 8.0 * p + 1.0);
  }
}

} // namespace mapping
} // namespace precice

#endif
