
#include <hip/hip_runtime.h>
#ifdef GINKGO_BUILD_CUDA
#include <ginkgo/ginkgo.hpp>
#include <cusolverDn.h>
#include <cublas_v2.h>
#include <cuda_runtime_api.h>
#include <cuda.h>
#include "device_launch_parameters.h"
#include <cuda_runtime.h>
#include "utils/Event.hpp"
#include "utils/EventUtils.hpp"

using GinkgoMatrix = gko::matrix::Dense<>;

// Handles for low-level CUDA libraries
cusolverDnHandle_t solverHandle;
cusolverStatus_t cusolverStatus = CUSOLVER_STATUS_SUCCESS;
cudaError_t cudaErrorCode = cudaSuccess;

// Important variables which track the state of the solver routines
double *dTau = nullptr;
void *dWork = nullptr;
void *hWork = nullptr;
int *devInfo = nullptr;

int cudaBackupDeviceId = 0;

void initQRSolver(const int deviceId=0){
    cudaGetDevice(&cudaBackupDeviceId);
    cudaSetDevice(deviceId);

    // Allocating important CUDA variables
    cudaMalloc((void **)&dWork, sizeof(double));
    cudaMalloc((void **)&devInfo, sizeof(int));

    cusolverDnCreate(&solverHandle);

}

void deInitQRSolver(){
    // Freeing CUDA variables
    cudaFree(dTau);
    cudaFree(dWork);
    cudaFree(devInfo);

    if (nullptr != solverHandle){
        cusolverDnDestroy(solverHandle);
    }
}

void computeQR(const std::shared_ptr<gko::Executor> &exec, GinkgoMatrix *A_Q, GinkgoMatrix *R)
{
    cusolverDnCreate(&solverHandle);

    // NOTE: It's important to transpose since cuSolver assumes column-major memory layout
    // Making a copy since every value will be overridden
    auto A_T = gko::share(GinkgoMatrix::create(exec, gko::dim<2>(A_Q->get_size()[1], A_Q->get_size()[0])));
    A_Q->transpose(gko::lend(A_T));

    // Setting dimensions for solver
    const unsigned int M = A_T->get_size()[1];
    const unsigned int N = A_T->get_size()[0];

    const int lda = max(1, M);
    const int k = min(M, N);

    size_t dLwork_geqrf = 0;
    size_t dLwork_orgqr = 0;
    size_t dLwork = 0;

    size_t hLwork_geqrf = 0;
    size_t hLwork = 0;

    cudaMalloc((void **)&dTau, sizeof(double) * M);

    precice::utils::Event calculateQRDecompEvent{"calculateQRDecomp"};

    // Query working space of geqrf and orgqr
    cusolverStatus = cusolverDnXgeqrf_bufferSize(solverHandle, nullptr, M, N, CUDA_R_64F, A_T->get_values(), lda, CUDA_R_64F, dTau, CUDA_R_64F, &dLwork_geqrf, &hLwork_geqrf);
    assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
    cusolverStatus = cusolverDnDorgqr_bufferSize(solverHandle, M, N, k, A_T->get_values(), lda, dTau, (int*) &dLwork_orgqr);
    assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
    dLwork = (dLwork_geqrf > dLwork_orgqr) ? dLwork_geqrf : dLwork_orgqr;
    cudaErrorCode = cudaMalloc((void **)&dWork, sizeof(double) * dLwork);
    assert(cudaSuccess == cudaErrorCode);

    // Compute QR factorization
    cusolverStatus = cusolverDnXgeqrf(solverHandle, nullptr, M, N, CUDA_R_64F, A_T->get_values(), lda, CUDA_R_64F, dTau, CUDA_R_64F, dWork, dLwork, hWork, hLwork, devInfo);
    cudaErrorCode = cudaDeviceSynchronize();
    assert(CUSOLVER_STATUS_SUCCESS == cusolverStatus);
    assert(cudaSuccess == cudaErrorCode);

    // Copy A_T to R s.t. the upper triangle corresponds to R
    A_T->transpose(gko::lend(R));

    // Compute Q
    cusolverStatus = cusolverDnDorgqr(solverHandle, M, N, k, A_T->get_values(), lda, dTau, (double*) dWork, dLwork, devInfo);
    cudaErrorCode = cudaDeviceSynchronize();
    assert(CUSOLVER_STATUS_SUCCESS == cusolverStatus);
    assert(cudaSuccess == cudaErrorCode);

    A_T->transpose(gko::lend(A_Q));

    cudaDeviceSynchronize();

    calculateQRDecompEvent.stop();

    cudaSetDevice(cudaBackupDeviceId); // Switch back to the GPU used for all coupled solvers

    return;
}
#endif
